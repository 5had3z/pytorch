
#include <hip/hip_runtime.h>
namespace fused_reduction {

// We have 6 dimensions, 3 in the grid, 3 in the block
// They can be 1 of 3 states,
// Reduction Domain - TEMPLATE STATE 0
//   - Participating in the reduction, has values coming in, one value coming
//     out across the dimension
// Iteration Domain - TEMPLATE STATE 1
//   - Not participating in the reduction, has values across the dimension after
//     the reduction
// Collapsed Domain - TEMPLATE STATE 2
//   - Previously reduced, doesn't need to be reduced on that dimension, doesn't
//     have values across that dimension
constexpr __device__ bool isReduce(int STATE) {
  return STATE == 0;
}

constexpr __device__ bool isIter(int STATE) {
  return STATE == 1;
}

constexpr __device__ bool isPred(int STATE) {
  return STATE == 2;
}

constexpr __device__ bool inactive(int STATE) {
  return STATE == 3;
}

constexpr __device__ bool activeNotIter(int STATE) {
  return STATE != 3 && STATE != 1;
}

} // namespace fused_reduction
